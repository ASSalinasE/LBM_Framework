#include <hip/hip_runtime.h>
#include "include/BC.cuh"
#include "include/utils.cuh"
#include "../include/macros.h"

__device__ void OBC(prec* localf, const prec* __restrict__ f, int i, int j, int Lx, int Ly){
	localf[j] = f[IDXcm(i, j, Lx, Ly)];
}

__device__ void BBBC(prec* localf, int j){
	int op[] = {3,4,1,2,7,8,5,6};
	localf[j] = localf[op[j-1]];
}

__device__ void SBC(prec* localf, int j, unsigned char b1, unsigned char b2){
	int op[] = {3,4,1,2,7,8,5,6};
	if(j < 5)
		localf[j] = localf[op[j-1]];
	else{
		int right[] = {5,6,7,4};
		int left[]  = {7,4,5,6};
		int index   = j-5;
		if (((b1>>(j-1) == b1>>(right[index])) && (b2>>(j-1) == b2>>(right[index]))) && 
			((b1>>(j-1) != b1>>(left[index] )) || (b2>>(j-1) != b2>>(left[index] ))))
			localf[j] = localf[left[index]+1];
		else if (((b1>>(j-1) == b1>>(left[index] )) && (b2>>(j-1) == b2>>(left[index] ))) && 
				 ((b1>>(j-1) != b1>>(right[index])) || (b2>>(j-1) != b2>>(right[index]))))
			localf[j] = localf[right[index]+1];
		else
			localf[j] = localf[op[j-1]];
	}
}

__device__ void PBC(prec* localf, const prec* __restrict__ f, int i, int j, 
					int Lx, int Ly, int* ex, int* ey){
	int y = i/Lx;
	int x = i - y * Lx;
	int xop = (Lx + x - ex[j])%Lx;
	int yop = (Ly + y - ey[j])%Ly;
	int iop = xop + yop * Lx;
	localf[j] = f[IDXcm(iop, j, Lx, Ly)];
}