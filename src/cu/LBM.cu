#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include "include/setup.cuh"
#include "include/LBMkernels.cuh"
#include "include/SWE.cuh"
#include "include/utils.cuh"
#include "../cpp/include/files.h"
#include "../include/structs.h"
#include "../include/macros.h"

void timeStep(configStruct config, mainStruct device, cudaStruct *deviceOnly, 
				 hipEvent_t ct1, hipEvent_t ct2, prec *msecs) {
	float dt;
	hipEventRecord(ct1);
	LBMpull <<<config.gridSize,config.blockSize>>> (config, device.b, deviceOnly->binary1, 
								deviceOnly->binary2, deviceOnly->f1, deviceOnly->f2, deviceOnly->h);
	pointerSwap(deviceOnly);
	hipEventRecord(ct2);
	hipEventSynchronize(ct2);
	hipEventElapsedTime(&dt, ct1, ct2);
	*msecs += dt;
}

void setup(configStruct config, mainStruct device, cudaStruct deviceOnly) {
	binaryKernel <<<config.gridSize,config.blockSize>>> (config, deviceOnly.binary1, deviceOnly.binary2);
	hKernel <<<config.gridSize,config.blockSize>>> (config, device.w, device.b, deviceOnly.h);
	fKernel <<<config.gridSize,config.blockSize>>> (config, deviceOnly.h, deviceOnly.f1);
}

void copyAndWriteResultData(configStruct config, mainStruct host, mainStruct device, cudaStruct deviceOnly, int t){
	wKernel <<<config.gridSize,config.blockSize>>> (config, deviceOnly.h, device.b, device.w);
	uint pBytes = config.Lx * config.Ly * sizeof(prec);
	hipMemcpy(host.w, device.w, pBytes, hipMemcpyDeviceToHost);
	writeOutput(config, t, host.w);
}

void LBM(configStruct config, mainStruct host, mainStruct device, cudaStruct *deviceOnly) {
	setup(config, device, *deviceOnly);

	int t = 0;
	hipEvent_t ct1, ct2;
	hipEventCreate(&ct1);
	hipEventCreate(&ct2);
	prec msecs = 0;
	std::cerr << std::fixed << std::setprecision(1);
	while (t <= config.timeMax) {
		t++;
		timeStep(config, device, deviceOnly, ct1, ct2, &msecs);
		if (config.dtOut != 0 && t%config.dtOut == 0) {
			std::cout << "Time step: " << t << " (" << 100.0*t / config.timeMax << "%)" << std::endl;
			copyAndWriteResultData(config, host, device, *deviceOnly, t);
		}
	}
	if (config.dtOut == 0) 
		copyAndWriteResultData(config, host, device, *deviceOnly, t);
	std::cout << "Average time per time step: " << msecs / config.timeMax << "[ms]" << std::endl;
}

