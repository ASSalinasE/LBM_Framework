#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "include/LBMkernels.cuh"
#include "include/utils.cuh"
#include "include/SWE.cuh"
#include "include/PDEfeq.cuh"
#include "include/BC.cuh"
#include "../include/structs.h"
#include "../include/macros.h"
 
__device__ void calculateMacroscopic(prec* localMacroscopic, prec* localf, prec e){
	localMacroscopic[0] = localf[0] + (localf[1] + localf[2] + localf[3] + localf[4]) + (localf[5] + localf[6] + localf[7] + localf[8]);
	localMacroscopic[1] = e * ((localf[1] - localf[3]) + (localf[5] - localf[6] - localf[7] + localf[8])) / localMacroscopic[0];
	localMacroscopic[2] = e * ((localf[2] - localf[4]) + (localf[5] + localf[6] - localf[7] - localf[8])) / localMacroscopic[0];
}

__global__ void LBMpull(const configStruct config,
	const prec* __restrict__ b, const unsigned char* __restrict__ binary1, 
	const unsigned char* __restrict__ binary2, const prec* __restrict__ f1, 
	prec* f2, prec* h) {
	int i = threadIdx.x + blockIdx.x*blockDim.x;	
	if (i < config.Lx*config.Ly) {
		unsigned char b1 = binary1[i];
		unsigned char b2 = binary2[i];
		if(b1 != 0 || b2 != 0){
			int ex[8] = {1,0,-1,0,1,-1,-1,1};		
			int ey[8] = {0,1,0,-1,1,1,-1,-1};
			prec forcing[8];
			#if PDE == 1
				calculateForcingSWE(forcing, h, b, config.e, i, config.Lx, ex, ey);
			#elif PDE == 5
				calculateForcingUser(forcing, h, b, config.e, i, config.Lx, ex, ey);
			#else 
				for (int j = 0; j < 8; j++)
					forcing[j] = 0;
			#endif

			prec localf[9];
			localf[0] = f1[i]; 
			for (int j = 1; j < 9; j++){
				if(((b1>>(j-1)) & 1) & (~(b2>>(j-1)) & 1)) 
					localf[j] = f1[IDXcm(IDX(i, j, config.Lx, ex, ey), j, config.Lx, config.Ly)] + forcing[j-1];
				else if((~(b1>>(j-1)) & 1) & (~(b2>>(j-1)) & 1)) 
					localf[j] = f1[IDXcm(i, j, config.Lx, config.Ly)];
			}

			for (int j = 1; j < 9; j++)
				if((~(b1>>(j-1)) & 1) & ((b2>>(j-1)) & 1)) 
					#if BC1 == 1
						OBC(localf, f1, i, j, config.Lx, config.Ly);
					#elif BC1 == 2
						PBC(localf, f1, i, j, config.Lx, config.Ly, ex, ey);
					#elif BC1 == 3
						BBBC(localf, j);
					#elif BC1 == 4
						SBC(localf, j, b1, b2);
					#elif BC1 == 5
						UBC1(localf, f1, i, j, config.Lx, config.Ly, ex, ey, b1, b2);
					#elif BC1 == 6
						UBC2(localf, f1, i, j, config.Lx, config.Ly, ex, ey, b1, b2);
					#endif

			#if BC2 != 0
			for (int j = 1; j < 9; j++)
				if(((b1>>(j-1)) & 1) & ((b2>>(j-1)) & 1)) 
					#if BC2 == 1
						localf[j] = OBC(localf, f1, i, j, config.Lx, config.Ly);
					#elif BC2 == 2
						localf[j] = PBC(localf, f1, i, j, config.Lx, config.Ly, ex, ey);
					#elif BC2 == 3
						localf[j] = BBBC(localf, j);
					#elif BC2 == 4
						localf[j] = SBC(localf, j, b1, b2);
					#elif BC2 == 5
						localf[j] = BC1User(localf, f1, i, j, config.Lx, config.Ly, ex, ey, b1, b2);
					#elif BC2 == 6
						localf[j] = BC2User(localf, f1, i, j, config.Lx, config.Ly, ex, ey, b1, b2);
					#endif
			#endif

			prec localMacroscopic[3];
			calculateMacroscopic(localMacroscopic, localf, config.e);
			h[i] = localMacroscopic[0];

			prec feq[9];
			#if PDE == 1
				calculateFeqSWE(feq, localMacroscopic, config.e);
			#elif PDE == 2
				calculateFeqHE(feq, localMacroscopic, config.e);
			#elif PDE == 3
				calculateFeqWE(feq, localMacroscopic, config.e);
			#elif PDE == 4
				calculateFeqNSE(feq, localMacroscopic, config.e);
			#elif PDE == 5
				calculateFeqUser(feq, localMacroscopic, config.e);
			#endif
			
			for (int j = 0; j < 9; j++)
				f2[IDXcm(i, j, config.Lx, config.Ly)] = localf[j] - (localf[j] - feq[j]) / config.tau;
		}
	} 
} 
