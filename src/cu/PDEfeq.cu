#include <hip/hip_runtime.h>
#include "include/PDEfeq.cuh"
#include "../include/macros.h"

__device__ void calculateFeqHE(prec* feq, prec* localMacroscopic, prec e){	
	prec factor = 1.0 / 9;	
	prec localT = localMacroscopic[0];

	feq[0] = localT * factor * 4;
	feq[1] = localT * factor;
	feq[2] = localT * factor;
	feq[3] = localT * factor;
	feq[4] = localT * factor;
	feq[5] = localT * factor * 0.25;
	feq[6] = localT * factor * 0.25;
	feq[7] = localT * factor * 0.25;
	feq[8] = localT * factor * 0.25;
}

__device__ void calculateFeqWE(prec* feq, prec* localMacroscopic, prec e){
  
}

__device__ void calculateFeqNSE(prec* feq, prec* localMacroscopic, prec e){
  prec factor = 1.0 / 9;	
	prec localrho = localMacroscopic[0];
	prec localux = localMacroscopic[1];
	prec localuy = localMacroscopic[2];
  
	prec usq = 1.5 * (localux * localux + localuy * localuy);
	prec ux3 = 3.0 * localux;
	prec uy3 = 3.0 * localuy;
	prec uxuy5 = ux3 + uy3;
	prec uxuy6 = uy3 - ux3;

	feq[0] = localrho * factor * 4 *    (1                                      - usq);
	feq[1] = localrho * factor *        (1 +   ux3 +     4.5 * ux3*ux3 * factor - usq);
	feq[2] = localrho * factor *        (1 +   uy3 +     4.5 * uy3*uy3 * factor - usq);
	feq[3] = localrho * factor *        (1 -   ux3 +     4.5 * ux3*ux3 * factor - usq);
	feq[4] = localrho * factor *        (1 -   uy3 +     4.5 * uy3*uy3 * factor - usq);
	feq[5] = localrho * factor * 0.25 * (1 + uxuy5 + 4.5 * uxuy5*uxuy5 * factor - usq);
	feq[6] = localrho * factor * 0.25 * (1 + uxuy6 + 4.5 * uxuy6*uxuy6 * factor - usq);
	feq[7] = localrho * factor * 0.25 * (1 - uxuy5 + 4.5 * uxuy5*uxuy5 * factor - usq);
	feq[8] = localrho * factor * 0.25 * (1 - uxuy6 + 4.5 * uxuy6*uxuy6 * factor - usq);
}
