#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "include/SWE.cuh"
#include "include/utils.cuh"
#include "../include/structs.h"
#include "../include/macros.h"

__device__ void calculateFeqSWE(prec* feq, prec* localMacroscopic, prec e){	
	prec factor = 1 / (9 * e*e);	
	prec localh = localMacroscopic[0];
	prec localux = localMacroscopic[1];
	prec localuy = localMacroscopic[2];
	prec gh  = 1.5 * 9.8 * localh;
	prec usq = 1.5 * (localux * localux + localuy * localuy);
	prec ux3 = 3.0 * e * localux;
	prec uy3 = 3.0 * e * localuy;
	prec uxuy5 = ux3 + uy3;
	prec uxuy6 = uy3 - ux3;

	feq[0] = localh * (1 - factor * (5.0 * gh + 4.0 * usq));
	feq[1] = localh * factor * (gh + ux3 + 4.5 * ux3*ux3 * factor - usq);
	feq[2] = localh * factor * (gh + uy3 + 4.5 * uy3*uy3 * factor - usq);
	feq[3] = localh * factor * (gh - ux3 + 4.5 * ux3*ux3 * factor - usq);
	feq[4] = localh * factor * (gh - uy3 + 4.5 * uy3*uy3 * factor - usq);
	feq[5] = localh * factor * 0.25 * (gh + uxuy5 + 4.5 * uxuy5*uxuy5 * factor - usq);
	feq[6] = localh * factor * 0.25 * (gh + uxuy6 + 4.5 * uxuy6*uxuy6 * factor - usq);
	feq[7] = localh * factor * 0.25 * (gh - uxuy5 + 4.5 * uxuy5*uxuy5 * factor - usq);
	feq[8] = localh * factor * 0.25 * (gh - uxuy6 + 4.5 * uxuy6*uxuy6 * factor - usq);
}

__device__ void calculateForcingSWE(prec* forcing, prec* h, const prec* __restrict__ b, prec e, 
									int i, int Lx, int* ex, int* ey){
	prec factor = 1 / (6 * e*e);
	prec localh = h[i];
	prec localb = b[i];
	for (int j = 0; j < 4; j++){
		int index = IDX(i, j, Lx, ex, ey);
		forcing[j] = factor * 9.8 * (localh + h[index]) * (b[index] - localb);
	}
	for (int j = 4; j < 8; j++){
		int index = IDX(i, j, Lx, ex, ey);
		forcing[j] = factor * 0.25 * 9.8 * (localh + h[index]) * (b[index] - localb);
	}
}

__global__ void hKernel(const configStruct config, const prec* __restrict__ w,
	const prec* __restrict__ b, prec* h){

	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i < config.Lx*config.Ly) {
		h[i] = w[i] - b[i];
	}
}

__global__ void wKernel(const configStruct config, const prec* __restrict__ h,
	const prec* __restrict__ b, prec* w){

	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i < config.Lx*config.Ly) {
		w[i] = h[i] + b[i];
	}
}
