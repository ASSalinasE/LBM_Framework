#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "include/setup.cuh"
#include "include/utils.cuh"
#include "include/SWE.cuh"
#include "../include/structs.h"
#include "../include/macros.h"

__global__ void binaryKernel(const configStruct config, 
	unsigned char* binary1, unsigned char* binary2) {

	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i < config.Lx*config.Ly) {
		unsigned char b1;
		unsigned char b2;
		int y = (int)i / config.Lx;
		int x = i - y * config.Lx;
		if (y == 0) {
			if (x == 0){
				b1 = 4 + 8 + 64;
				b2 = 1 + 2 + 16;
			}
			else if (x == config.Lx - 1){
				b1 = 1 + 8 + 128;
				b2 = 2 + 4 + 32;
			}
			else{
				b1 = 1 + 4 + 8 + 64 + 128;
				b2 = 2 + 16 + 32; 
			}
		}
		else if (y == config.Ly - 1) {
			if (x == 0) {
				b1 = 2 + 4 + 32;
				b2 = 1 + 8 + 128;
			}
			else if (x == config.Lx - 1){ 
				b1 = 1 + 2 + 16;
				b2 = 4 + 8 + 64;
			}
			else{ 
				b1 = 1 + 2 + 4 + 16 + 32;
				b2 = 8 + 64 + 128;
			}
		}
		else {
			if (x == 0){
				b1 = 2 + 4 + 8 + 32 + 64;
				b2 = 1 + 16 + 128;
			}
			else if (x == config.Lx - 1){
				b1 = 1 + 2 + 8 + 16 + 128;
				b2 = 4 + 32 + 64;
			}
			else{
				b1 = 255;
				b2 = 0;
			}
		}
		binary1[i] = b1;
		binary2[i] = b2;
	}
}

__global__ void fKernel(const configStruct config,
	const prec* __restrict__ h, prec* f) {

	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i < config.Lx*config.Ly) {
		prec feq[9];
		prec localMacroscopic[] = {h[i], 0, 0};
		#if PDE == 1
			calculateFeqSWE(feq, localMacroscopic, config.e);
		#elif PDE == 2
			calculateFeqHE(feq, localMacroscopic, config.e);
		#elif PDE == 3
			calculateFeqWE(feq, localMacroscopic, config.e);
		#elif PDE == 4
			calculateFeqNSE(feq, localMacroscopic, config.e);
		#elif PDE == 5
			calculateFeqUser(feq, localMacroscopic, config.e);
		#endif
		for (int j = 0; j < 9; j++)
			f[IDXcm(i, j, config.Lx, config.Ly)] = feq[j];
	}
}