#include "hip/hip_runtime.h"
#include <iomanip>
#include <stdio.h>
#include <iostream>
#include <time.h>
#include <hip/hip_runtime.h>
#include "include/structs.h"
#include "include/macros.h"
#include "cpp/include/input.h"
#include "cpp/include/config.h"
#include "cpp/include/output.h"
#include "cu/include/LBM.cuh"
#include "cu/include/utils.cuh"

int main(int argc, char* argv[]) {
	clock_t t1, t2; 
	t1 = clock();
	mainStruct host;
	mainStruct device;
	cudaStruct deviceOnly;
	configStruct config;

	setConfig(&config, argv, argc);
	createOutputDir(&config);
	readInput(&config, &host);
	writeConfig(config);
	writeOutput(config, 0, host.w);
	memoryInit(config, &deviceOnly, &device, host);

	std::cout << "Starting LBM loop" << std::endl;
	LBM(config, host, device, &deviceOnly);

	memoryFree(host, device, deviceOnly);

	t2 = clock();
	prec elapsedTime = 1000.0 * (prec)(t2 - t1) / CLOCKS_PER_SEC;
	std::cout << "Program successfully terminated\n"
			  << "Total execution time: " << elapsedTime << "[ms]" << std::endl;
	exit(EXIT_SUCCESS);
} 

